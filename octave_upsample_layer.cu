#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/octave_upsample_layer.hpp"

namespace caffe {
template <typename Dtype>
__global__ void OctaveupsampleForward(const int nthreads, const Dtype* const bottom_data,
	const int num, const int channels,
	const int bottom_height, const int bottom_width,
	const int upsample_height, const int upsample_width,
	const int copy_height, const int copy_width,
	Dtype* const top_data)
  {
    CUDA_KERNEL_LOOP(index, nthreads) {
	const int bw = index % bottom_width;
	const int bh = (index / bottom_width) % bottom_height;
	const int c = (index / bottom_width / bottom_height) % channels;
	const int n = index / bottom_width / bottom_height / channels;
	 //bottom index = index
	 int idx_b = index;
	 //caulculate bottom_slice
	 //const Dtype* const bottom_slice =
	 //    bottom_data + (n * channels + c) * bottom_height * bottom_width;
	 //caulculate top_slice
	  //Dtype*  top_slice = 
		 //top_data + (n * channels + c) * upsample_height * upsample_width;
	 for (int ch = 0; ch < copy_height; ++ch) {
		 for (int cw = 0; cw < copy_width; ++cw) {
			 //caulculate index in top
			 int idx_t =  (n * channels + c) * upsample_height * upsample_width +
				              bh*upsample_width*copy_height + bw* copy_width + ch*upsample_width + cw;
			 top_data[idx_t] = bottom_data[idx_b];
		 }
	  }
    }
  }

template <typename Dtype>
void OctaveUpsampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_set(top[0]->count(), Dtype(0), top_data);
  int bottom_count = bottom[0]->count();
  OctaveupsampleForward<Dtype> << <CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS >> >(
	  bottom_count, bottom_data, bottom[0]->num(), channels_,
	  height_, width_,
	  upsample_h_, upsample_w_,
      copy_w, copy_h
	  , top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void OctaveupsampleBackward(const int nthreads, const Dtype* const  top_diff,
	const int num, const int channels,
	const int upsample_height, const int upsample_width,
	const int copy_height, const int copy_width,
	const int bottom_height, const int bottom_width,
	Dtype* const bottom_diff)
{
    CUDA_KERNEL_LOOP(index, nthreads) {
		const int bw = index % bottom_width;
		const int bh = (index / bottom_width) % bottom_height;
		const int c = (index / bottom_width / bottom_height) % channels;
		const int n = index / bottom_width / bottom_height / channels;
		//bottom index = index
		int idx_b = index;
		//caulculate bottom_slice
		 //Dtype*  bottom_slice =
			//bottom_diff + (n * channels + c) * bottom_height * bottom_width;
		//caulculate top_slice
		//const Dtype* const top_slice =
		//	top_diff + (n * channels + c) * upsample_height * upsample_width;
		for (int ch = 0; ch < copy_height; ++ch) {
			for (int cw = 0; cw < copy_width; ++cw) {
				//caulculate index in top
				int idx_t = (n * channels + c) * upsample_height * upsample_width  +
					             bh*upsample_width*copy_height + bw* copy_width + ch*upsample_width + cw;
				bottom_diff[idx_b] += top_diff[idx_t];
			}
		}
    }
  }

template <typename Dtype>
void OctaveUpsampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    caffe_gpu_set(bottom_count, Dtype(0.), bottom_diff);
	OctaveupsampleBackward<Dtype> << <CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS >> >(
		bottom_count, top_diff, bottom[0]->num(), channels_,
		upsample_h_, upsample_w_,
		copy_w, copy_h,
		height_, width_,
		bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}
INSTANTIATE_LAYER_GPU_FUNCS(OctaveUpsampleLayer);
}  // namespace caffe
